#include <stdio.h>
#include "cuda-helper.h"

//--------------------------------------------------------------------------
// functions to help diagnose cuBLAS failures.
const char *cublasGetErrorString(hipblasStatus_t status) {
  switch(status) {
  case HIPBLAS_STATUS_SUCCESS:
    return "HIPBLAS_STATUS_SUCCESS";
  case HIPBLAS_STATUS_NOT_INITIALIZED:
    return "HIPBLAS_STATUS_NOT_INITIALIZED";
  case HIPBLAS_STATUS_ALLOC_FAILED:
    return "HIPBLAS_STATUS_ALLOC_FAILED";
  case HIPBLAS_STATUS_INVALID_VALUE:
    return "HIPBLAS_STATUS_INVALID_VALUE";
  case HIPBLAS_STATUS_ARCH_MISMATCH:
    return "HIPBLAS_STATUS_ARCH_MISMATCH";
  case HIPBLAS_STATUS_MAPPING_ERROR:
    return "HIPBLAS_STATUS_MAPPING_ERROR";
  case HIPBLAS_STATUS_EXECUTION_FAILED:
    return "HIPBLAS_STATUS_EXECUTION_FAILED";
  case HIPBLAS_STATUS_INTERNAL_ERROR:
    return "HIPBLAS_STATUS_INTERNAL_ERROR";
  case HIPBLAS_STATUS_NOT_SUPPORTED:
    return "HIPBLAS_STATUS_NOT_SUPPORTED";
  case HIPBLAS_STATUS_UNKNOWN:
    return "HIPBLAS_STATUS_UNKNOWN";
  default:
    return "UNKNOWN!!!";
  }
}

void _cublasTry(hipblasStatus_t hipblasStatus_t, 
	       const char *fileName, int lineNumber) {
  if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "%s in %s line %d\n",
	    cublasGetErrorString(hipblasStatus_t), fileName, lineNumber);
    exit(EXIT_FAILURE);
  }
}

//--------------------------------------------------------------------------
// function to help diagnose CUDA failures.

void _cudaTry(hipError_t cudaStatus, const char *fileName, int lineNumber) {
  if(cudaStatus != hipSuccess) {
    fprintf(stderr, "%s in %s line %d\n",
        hipGetErrorString(cudaStatus), fileName, lineNumber);
    exit(1);
  }
}
